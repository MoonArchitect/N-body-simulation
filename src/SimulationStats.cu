#include "hip/hip_runtime.h"
#include <tuple>
#include "NbodySystem.h"

#define BLOCK_SIZE 128


std::tuple<double, double, double> SimulationStats::computeLinearMomentum(Space space, float4* pos, float4* vel, int n) {
	double* dm; hipMalloc(&dm, sizeof(double)); hipMemset(dm, 0, sizeof(double));
	double* dkE; hipMalloc(&dkE, sizeof(double)); hipMemset(dkE, 0, sizeof(double));
	
	if(space == R2)
		SimulationStats::compute_LMoment_kE_R2Kernel << < 64, BLOCK_SIZE >> > (dm, dkE, pos, vel, n);
	else
		SimulationStats::compute_LMoment_kE_R3Kernel << < 64, BLOCK_SIZE >> > (dm, dkE, pos, vel, n);
	hipDeviceSynchronize();

	double lmoment; hipMemcpy(&lmoment, dm, sizeof(double), hipMemcpyDeviceToHost);
	double kE; hipMemcpy(&kE, dkE, sizeof(double), hipMemcpyDeviceToHost);
	return { lmoment, kE, -1 };
}

__global__ void SimulationStats::compute_LMoment_kE_R2Kernel(double* momentum, double* kE, float4* pos, float4* vel, int n) {
	int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	int stride = BLOCK_SIZE * gridDim.x;
	double lm = 0, lkE = 0;

	while (i < n) {
		double v = sqrt(vel[i].x * vel[i].x + vel[i].y * vel[i].y);
		lm += v * pos[i].w;
		lkE += 0.5 * v * v * pos[i].w;

		i += stride;
	}

	atomicAdd(momentum, lm);
	atomicAdd(kE, lkE);
}

__global__ void SimulationStats::compute_LMoment_kE_R3Kernel(double* momentum, double* kE, float4* pos, float4* vel, int n) {
	int i = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	int stride = BLOCK_SIZE * gridDim.x;
	double lm = 0, lkE = 0;

	while (i < n) {
		double v = sqrt(vel[i].x * vel[i].x + vel[i].y * vel[i].y + vel[i].z * vel[i].z);
		lm += v * pos[i].w;
		lkE += 0.5 * v * v * pos[i].w;

		i += stride;
	}

	atomicAdd(momentum, lm);
	atomicAdd(kE, lkE);
}


__global__ void SimulationStats::compute_pE_Kernel(double* pE, float4* pos, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) {
		float4 b = pos[i];
		double lpE = 0;

		for (unsigned int tile = 0; tile < gridDim.x; tile++) {
			__shared__ float4 spos[BLOCK_SIZE];
			spos[threadIdx.x] = pos[tile * blockDim.x + threadIdx.x];

			__syncthreads();

#pragma unroll
			for (int j = 0; j < BLOCK_SIZE; j++) {
				float dx = spos[j].x - b.x;
				float dy = spos[j].y - b.y;
				float distSqr = dx * dx + dy * dy;
				float invDist = rsqrtf(distSqr);
				float invDist3 = invDist * invDist * spos[j].w;
			}
		}

		atomicAdd(pE, lpE);
	}
}
