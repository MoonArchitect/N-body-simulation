#include "hip/hip_runtime.h"
#include <random>
#include <math.h>
#include "NbodySystem.h"
#include <iostream>

using namespace InitialConditions;

///////////////////////////////////////// Debug Version /////////////////////////////////////////

Standard::Standard(float2 pos, float2 vel, float2 size, float2 massRange, float2 velRange)
	: pos(pos), vel(vel), size(size), massRange(massRange), velRange(velRange) {}

void Standard::initialize(int offset, int n, NbodySystem* system) {
	float a = 1.0;
	float pi = 3.14159265;
	std::default_random_engine generator((unsigned int)(234234234));
	//std::uniform_real_distribution<float> distribution(1300, 40000);
	std::exponential_distribution<float> distribution(6);
	std::uniform_real_distribution<float> distribution_theta(0.0, 2 * pi);
	float* thetaArray = new float[n];
	for (int i = 0; i < n; i++) {
		float theta = distribution_theta(generator);
		thetaArray[i] = theta;
		float r = (distribution(generator) + 0.02) * 50000;
		float offsetX = 0;
		float offsetY = 0;
		if (offset == 0) {
			offsetX = 0;
			offsetY = 0;
		}
		if (i == 0) {
			system->host.pos_mass[i + offset].w = 1000000;
			system->host.pos_mass[i + offset].x = offsetX;
			system->host.pos_mass[i + offset].y = offsetY;
			system->host.pos_mass[i + offset].z = 0;
		}
		else {
			system->host.pos_mass[i + offset].w = 100;
			system->host.pos_mass[i + offset].x = system->host.pos_mass[offset].x + r * cos(theta);
			system->host.pos_mass[i + offset].y = system->host.pos_mass[offset].y + r * sin(theta);
			system->host.pos_mass[i + offset].z = sinf(r / 2000) * 2000;
		}
	}

	system->updateDeviceData();
	system->computeAcceleration(true);
	system->updateHostData();
	
	for (int i = 0; i < n; i++) {
		float rotation = offset == 0 ? 1 : -1; 
		float dx = system->host.pos_mass[0].x - system->host.pos_mass[i].x;
		float dy = system->host.pos_mass[0].y - system->host.pos_mass[i].y;
		float dz = system->host.pos_mass[0].z - system->host.pos_mass[i].z; 
		float dist = sqrtf(dx * dx + dy * dy + dz * dz + 1);
		float Fx = system->host.acc[i + offset].x;
		float Fy = system->host.acc[i + offset].y;
		float Fz = system->host.acc[i + offset].z;
		float F = sqrtf(Fx * Fx + Fy * Fy + Fz * Fz);
		float v = sqrtf(dist * F);
		
		
		if (i == 0) {
			system->host.vel[i + offset].x = 0;
			system->host.vel[i + offset].y = 0;
			system->host.vel[i + offset].z = 0;
		}
		else {
			system->host.vel[i + offset].x = rotation * v * sin(thetaArray[i]);
			system->host.vel[i + offset].y = -rotation * v * cos(thetaArray[i]);
			system->host.vel[i + offset].z = 5;
		}
	}




	
	//float max_r = 40000.0f;
	//std::default_random_engine generator((unsigned int)(32423462));
	//std::uniform_real_distribution<float> dist_u(-1, 1);
	//std::uniform_real_distribution<float> dist_lambda(0, 1);
	//std::uniform_real_distribution<float> dist_phi(0.0f, 2 * 3.14159265f);
	//for (int i = 0; i < n; i++) {
	//	system->host.pos_mass[i].w = 1000;
	//
	//	float phi = dist_phi(generator);
	//	float lambda = dist_lambda(generator);
	//	float u = dist_u(generator);
	//
	//	system->host.pos_mass[i].x = (max_r * powf(lambda, 1.0f / 3.0f) * sqrtf(1 - u * u) * cosf(phi));
	//	system->host.pos_mass[i].y = (max_r * powf(lambda, 1.0f / 3.0f) * sqrtf(1 - u * u) * sinf(phi));
	//	system->host.pos_mass[i].z = 0;
	//	system->host.vel[i].x = (rand() / (float)RAND_MAX - 0.5) * 10;
	//	system->host.vel[i].y = (rand() / (float)RAND_MAX - 0.5) * 10;
	//	system->host.vel[i].z = 0;
	//}
	//
	system->updateDeviceData();
}

///////////////////////////////////////// UniformBox /////////////////////////////////////////

UniformBox::UniformBox(float2 pos, float2 vel, float2 size, float2 massRange, float2 velRange) 
	: pos(pos), vel(vel), size(size), massRange(massRange), velRange(velRange) {}

void UniformBox::initialize(int offset, int n, NbodySystem *system) {
	std::default_random_engine generator((unsigned int)(32423462));
	std::uniform_real_distribution<float> x_dist(pos.x - size.x / 2, pos.x + size.x / 2);
	std::uniform_real_distribution<float> y_dist(pos.y - size.y / 2, pos.y + size.y / 2);
	std::uniform_real_distribution<float> z_dist(pos.y - size.y / 2, pos.y + size.y / 2);
	std::uniform_real_distribution<float> mass_dist(massRange.x, massRange.y);
	std::uniform_real_distribution<float> vel_dist(velRange.x, velRange.y);
	std::uniform_real_distribution<float> phi_dist(0.0f, 2 * 3.14159265f);
	
	for (int i = 0; i < n; i++) {
		float x = x_dist(generator);
		float y = y_dist(generator);
		//float z = z_dist(generator);
		float mass = mass_dist(generator);
		float phi = phi_dist(generator);
		float v = vel_dist(generator);
		
		system->host.pos_mass[offset + i] = make_float4(x, y, 0, mass);
		system->host.vel[offset + i] = make_float4(v * cosf(phi) + vel.x, v * sinf(phi) + vel.y, 0, 0);
	}

	system->updateDeviceData();
}

///////////////////////////////////////////// UniformEllipsoid /////////////////////////////////////////////

UniformEllipsoid::UniformEllipsoid(float2 pos, float2 vel, float2 radius, float2 massRange, float2 velRange)
	: pos(pos), vel(vel), radius(radius), massRange(massRange), velRange(velRange) {}


void UniformEllipsoid::initialize(int offset, int n, NbodySystem* system) {
	std::default_random_engine generator((unsigned int)(32423462));
	std::uniform_real_distribution<float> u_dist(-1, 1);
	std::uniform_real_distribution<float> lambda_dist(0.1, 1);
	std::uniform_real_distribution<float> phi_dist(0.0f, 2 * 3.14159265f);
	std::uniform_real_distribution<float> mass_dist(massRange.x, massRange.y);
	std::uniform_real_distribution<float> vel_dist(velRange.x, velRange.y);
	
	for (int i = 0; i < n; i++) {

		float phi = phi_dist(generator);
		float lambda = lambda_dist(generator);
		float u = u_dist(generator);
		float mass = mass_dist(generator);
		float v = vel_dist(generator);


		system->host.pos_mass[i] = make_float4(
			pos.x + radius.x * cosf(phi) * lambda, //powf(lambda, 1.0f / 3.0f) * sqrtf(1 - u * u) * cosf(phi),
			pos.y + radius.y * sinf(phi) * lambda, //powf(lambda, 1.0f / 3.0f) * sqrtf(1 - u * u) * sinf(phi),
			0,
			mass
		);

		system->host.vel[i] = make_float4(
			v * cosf(phi) + vel.x,
			v * sinf(phi) + vel.y,
			0,
			0
		);
	}

	system->updateDeviceData();
}


///////////////////////////////////////// DiskModel //////////////////////////////////////////

DiskModel::DiskModel(float2 pos, float2 vel, float2 radius, float2 massRange, float2 velRange) 
	: UniformEllipsoid::UniformEllipsoid(pos, vel, radius, massRange, velRange) {}


void DiskModel::initialize(int offset, int n, NbodySystem* system) {
	
	UniformEllipsoid::initialize(offset, n, system);
	
	system->host.pos_mass[0] = { pos.x, pos.y, 0, 10000000 };
	system->updateDeviceData();

	system->computeAcceleration(true);

	system->updateHostData();

	for (int i = 0; i < n; i++) {
		float x = system->host.pos_mass[i].x;
		float y = system->host.pos_mass[i].y;
		float z = system->host.pos_mass[i].z;

		float r = sqrtf(x * x + y * y + z * z + 0.0001f);
		float a = sqrtf(system->host.acc[i].x * system->host.acc[i].x + system->host.acc[i].y * system->host.acc[i].y + system->host.acc[i].z * system->host.acc[i].z);

		float v = sqrtf(a * r);
		
		float vx = v * x / r;
		float vy = v * y / r;
		float vz = v * z / r;

		system->host.vel[i] = make_float4(
			-vy,
			vx,
			i == 0 ? 0 : 15,//vz,
			0
		);
	}

	system->updateDeviceData();
}
